#include "hip/hip_runtime.h"
/***************************************************************************
 *  Copyright 2014, 2017, 2018, 2020 Marcelo Y. Matuda                     *
 *                                                                         *
 *  This program is free software: you can redistribute it and/or modify   *
 *  it under the terms of the GNU General Public License as published by   *
 *  the Free Software Foundation, either version 3 of the License, or      *
 *  (at your option) any later version.                                    *
 *                                                                         *
 *  This program is distributed in the hope that it will be useful,        *
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of         *
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the          *
 *  GNU General Public License for more details.                           *
 *                                                                         *
 *  You should have received a copy of the GNU General Public License      *
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.  *
 ***************************************************************************/

#include "VectorialCombinedTwoMediumImagingCUDAProcessor2.h"

#include <algorithm> /* copy */
#include <cmath> /* ceil, sqrt */
#include <cstring> /* memset */

#include <tbb/partitioner.h>
#include <tbb/tbb.h>

#include "hip/hip_runtime.h"

#include "ArrayGeometry.h"
#include "Exception.h"
#include "FermatPrinciple.h"
#include "Geometry.h"
#include "Log.h"
#include "Timer.h"
#include "Util.h"

#include "CUDAUtil.h"



// Depends on the signal.
// 1.0 --> pi radian / sample at the original sampling rate.
#define UPSAMP_FILTER_HALF_TRANSITION_WIDTH (0.2)

#define NUM_RAW_DATA_BUFFERS 2
#define USE_TRANSPOSE 1

#define BLOCK_SIZE 64

#ifndef MFloat
# define MFloat float
#endif



namespace Lab {

// NVIDIA sm_12:
//   - Local (shared) memory has 16 banks.

#define TRANSP_BLOCK_SIZE 16
#define NUM_RX_ELEM 32

extern __shared__ float sharedArray[];

// Defined in VectorialCombinedTwoMediumImagingCUDAProcessor.cu.
extern __global__ void transposeKernel(float* rawData, float* rawDataT, int oldSizeX, int oldSizeY);
extern __global__ void processImageKernel(float* rawData, int numGridPoints, float* gridValueRe,
						float* gridValueIm, float* rxApod);
extern __global__ void processImagePCFKernel(float* rawData, int numGridPoints, float* gridValueRe,
						float* gridValueIm, float* rxApod, float pcfFactor);

//=============================================================================

struct VectorialCombinedTwoMediumImagingCUDAProcessor2Data {
	bool cudaDataInitialized;
	MFloat* rxApodDev;
	std::vector<MFloat*> rawDataList;
	std::vector<MFloat*> rawDataDevList;
#ifdef USE_TRANSPOSE
	MFloat* rawDataTDev;
#endif
	MFloat* gridValueRe;
	MFloat* gridValueReDev;
	MFloat* gridValueIm;
	MFloat* gridValueImDev;

	VectorialCombinedTwoMediumImagingCUDAProcessor2Data()
		: cudaDataInitialized()
		, rxApodDev()
		, rawDataList(NUM_RAW_DATA_BUFFERS)
		, rawDataDevList(NUM_RAW_DATA_BUFFERS)
#ifdef USE_TRANSPOSE
		, rawDataTDev()
#endif
		, gridValueRe()
		, gridValueReDev()
		, gridValueIm()
		, gridValueImDev()
	{}
};

template<typename TFloat>
struct CalculateDelays2 {
	void operator()(const tbb::blocked_range<unsigned int>& r) const {
		//LOG_DEBUG << "col = " << r.begin() << " n = " << (r.end() - r.begin());

		for (unsigned int col = r.begin(); col != r.end(); ++col) {
			if (minRowIdx[col] >= numRows) continue;

			for (unsigned int elem = 0; elem < config.numElementsMux; ++elem) {
				unsigned int lastInterfaceIdx = 0;

				// The first row above the interface.
				{
					const auto& point = gridXZ(col, minRowIdx[col]);

					// Fermat's principle. Find the fastest path.
					TFloat tMin;
					unsigned int idxMin;
					FermatPrinciple::findMinTimeInTwoSteps(
							fermatBlockSize,
							config.propagationSpeed1, config.propagationSpeed2,
							interfacePointList,
							xArray[elem], TFloat(0), point.x, point.z,
							tMin, idxMin);
					delayTensor(col, minRowIdx[col], elem) = tMin * fs;
					lastInterfaceIdx = idxMin;
				}

				const TFloat* medium1Delays = &medium1DelayMatrix(elem, 0);

				for (unsigned int row = minRowIdx[col] + 1; row < numRows; ++row) {
					const auto& point = gridXZ(col, row);
					unsigned int idxMin = lastInterfaceIdx;
					TFloat tC2Min;
					{
						const XZ<TFloat>& ifPoint = interfacePointList[idxMin];
						tC2Min = medium1Delays[idxMin] + Geometry::distance2D(ifPoint.x, ifPoint.z, point.x, point.z);
					}
					for (unsigned int idxSearch = idxMin + 1, end = interfacePointList.size(); idxSearch < end; ++idxSearch) {
						const XZ<TFloat>& ifPoint = interfacePointList[idxSearch];
						const TFloat tC2 = medium1Delays[idxSearch] + Geometry::distance2D(ifPoint.x, ifPoint.z, point.x, point.z);
						if (tC2 >= tC2Min) {
							break;
						} else {
							tC2Min = tC2;
							idxMin = idxSearch;
						}
					}
					if (idxMin == lastInterfaceIdx) { // if the previous search was not successful
						for (int idxSearch = static_cast<int>(idxMin) - 1; idxSearch >= 0; --idxSearch) { // if idxMin = 0, idxSearch will start with -1
							const XZ<TFloat>& ifPoint = interfacePointList[idxSearch];
							const TFloat tC2 = medium1Delays[idxSearch] + Geometry::distance2D(ifPoint.x, ifPoint.z, point.x, point.z);
							if (tC2 >= tC2Min) {
								break;
							} else {
								tC2Min = tC2;
								idxMin = idxSearch;
							}
						}
					}

//					unsigned int diff = (idxMin > lastInterfaceIdx) ?
//								idxMin - lastInterfaceIdx :
//								lastInterfaceIdx - idxMin;
//					if (diff > 1) {
//						LOG_DEBUG << "########## DIFF " << diff << " idxMin: " << idxMin << " col: " << col << " row - minRowIdx[col]: " << row - minRowIdx[col];
//					}

					delayTensor(col, row, elem) = tC2Min * fsInvC2;
					lastInterfaceIdx = idxMin;
				}

			}
		}
	}

	// Use only small types or references/pointers, because this object may be copied many times.
	const std::size_t numRows;
	const TwoMediumSTAConfiguration<TFloat>& config;
	const TFloat fs;
	const TFloat fsInvC2;
	const TFloat invC1;
	const TFloat invC2;
	const unsigned int fermatBlockSize;
	const std::vector<XZ<TFloat>>& interfacePointList;
	const std::vector<TFloat, tbb::cache_aligned_allocator<TFloat>>& xArray;
	const std::vector<unsigned int, tbb::cache_aligned_allocator<unsigned int>>& minRowIdx;
	const Matrix<TFloat, tbb::cache_aligned_allocator<TFloat>>& medium1DelayMatrix;
	const Matrix<XZ<TFloat>>& gridXZ;
	Tensor3<TFloat, tbb::cache_aligned_allocator<TFloat>>& delayTensor;
};

template<typename TFloat>
struct PrepareDataWithOneTxElem2 {
	void operator()(const tbb::blocked_range<unsigned int>& r) const {
		typename VectorialCombinedTwoMediumImagingCUDAProcessor2::PrepareDataThreadData<TFloat>& local = prepareDataTLS.local();

		for (unsigned int rxElem = r.begin(); rxElem != r.end(); ++rxElem) {
			if (upsamplingFactor > 1) {
				// Interpolate the signal.
				local.interpolator.interpolate(&acqDataList[baseElementIdx](rxElem, 0), samplesPerChannelLow, local.signal.data());
			} else {
				auto range = acqDataList[baseElementIdx].range2(rxElem);
				std::copy(range.begin(), range.end(), local.signal.begin());
			}

			Util::removeDC(local.signal.data(), local.signal.size());

			// Obtain the analytic signal.
			local.envelope.getAnalyticSignal(
					local.signal.data(),
					local.signal.size(),
					&signalTensor(stepIdx, rxElem, 0));
		}
	}

	// Use only small types or references/pointers, because this object may be copied many times.
	const std::size_t samplesPerChannelLow;
	const std::vector<Matrix<TFloat>>& acqDataList;
	const unsigned int upsamplingFactor;
	const unsigned int stepIdx;
	const unsigned int baseElementIdx;
	tbb::enumerable_thread_specific<typename VectorialCombinedTwoMediumImagingCUDAProcessor2::PrepareDataThreadData<TFloat>>& prepareDataTLS;
	Tensor3<std::complex<TFloat>, tbb::cache_aligned_allocator<std::complex<TFloat>>>& signalTensor;
};

template<typename TFloat>
struct ProcessColumnWithOneTxElem2 {
	void operator()(const tbb::blocked_range<unsigned int>& r) const {
		//LOG_DEBUG << "col = " << r.begin() << " n = " << (r.end() - r.begin());

		const unsigned int signalLength = signalTensor.n3();
		const unsigned int maxPosition = signalLength - 2;

		for (unsigned int col = r.begin(); col != r.end(); ++col) {
			unsigned int gridPointIdx = firstGridPointIdx[col] - firstGridPointIdx[firstCol];
			for (unsigned int row = minRowIdx[col]; row < numRows; ++row, ++gridPointIdx) {
				const TFloat* delays = &delayTensor(col, row, stepConfig.baseElem);
				const TFloat txDelay = delays[stepConfig.txElem];
				const TFloat txOffset = signalOffset + txDelay;
				const auto* p = &signalTensor(stepConfig.baseElemIdx, 0 /* rxElem */, 0);
				for (unsigned int rxElem = 0; rxElem < config.numElements; ++rxElem, p += signalLength) {
					const unsigned int rxIdx = rxElem * 2;

					// Linear interpolation.
					const TFloat position = txOffset + delays[rxElem];
					if (position >= 0.0f) {
						const unsigned int positionIdx = static_cast<unsigned int>(position);
						if (positionIdx <= maxPosition) {
							const TFloat k = position - positionIdx;
							const auto v0 = p[positionIdx];
							const auto v1 = p[positionIdx + 1];
							const std::complex<TFloat> v = v0 + k * (v1 - v0);
#ifdef USE_TRANSPOSE
							rawData[gridPointIdx * rawDataN2 + rxIdx    ] = v.real();
							rawData[gridPointIdx * rawDataN2 + rxIdx + 1] = v.imag();
#else
							rawData[ rxIdx      * rawDataN2 + gridPointIdx] = v.real();
							rawData[(rxIdx + 1) * rawDataN2 + gridPointIdx] = v.imag();
#endif
						} else {
#ifdef USE_TRANSPOSE
							rawData[gridPointIdx * rawDataN2 + rxIdx    ] = 0;
							rawData[gridPointIdx * rawDataN2 + rxIdx + 1] = 0;
#else
							rawData[ rxIdx      * rawDataN2 + gridPointIdx] = 0;
							rawData[(rxIdx + 1) * rawDataN2 + gridPointIdx] = 0;
#endif
						}
					} else {
#ifdef USE_TRANSPOSE
						rawData[gridPointIdx * rawDataN2 + rxIdx    ] = 0;
						rawData[gridPointIdx * rawDataN2 + rxIdx + 1] = 0;
#else
						rawData[ rxIdx      * rawDataN2 + gridPointIdx] = 0;
						rawData[(rxIdx + 1) * rawDataN2 + gridPointIdx] = 0;
#endif
					}
				}
			}
		}
	}

	// Use only small types or references/pointers, because this object may be copied many times.
	const unsigned int numRows;
	const unsigned int firstCol;
	const TwoMediumSTAConfiguration<TFloat>& config;
	const TFloat signalOffset;
	const Tensor3<std::complex<TFloat>, tbb::cache_aligned_allocator<std::complex<TFloat>>>& signalTensor;
	const VectorialCombinedTwoMediumImagingCUDAProcessor2::StepConfiguration stepConfig;
	const std::vector<unsigned int, tbb::cache_aligned_allocator<unsigned int>>& minRowIdx;
	const std::vector<unsigned int, tbb::cache_aligned_allocator<unsigned int>>& firstGridPointIdx;
	const Tensor3<TFloat, tbb::cache_aligned_allocator<TFloat>>& delayTensor;
	TFloat* rawData;
	unsigned int rawDataN2;
};

VectorialCombinedTwoMediumImagingCUDAProcessor2::VectorialCombinedTwoMediumImagingCUDAProcessor2(
			const TwoMediumSTAConfiguration<MFloat>& config,
			std::vector<Matrix<MFloat>>& acqDataList,
			unsigned int upsamplingFactor,
			AnalyticSignalCoherenceFactorProcessor<MFloat>& coherenceFactor,
			MFloat maxFermatBlockSize,
			MFloat peakOffset,
			unsigned int signalStartOffset)
		: config_(config)
		, acqDataList_(acqDataList)
		, upsamplingFactor_(upsamplingFactor)
		, coherenceFactor_(coherenceFactor)
		, maxFermatBlockSize_(maxFermatBlockSize)
		, lambda2_(config_.propagationSpeed2 / config_.centerFrequency)
		, rawDataN1_()
		, rawDataN2_()
		, rawDataSizeInBytes_()
{
	if (NUM_RAW_DATA_BUFFERS < 1) {
		THROW_EXCEPTION(InvalidValueException, "The number of raw data buffers must be >= 1.");
	}

	const std::size_t origSignalLength = acqDataList_[0].n2();

	signalOffset_ = (config_.samplingFrequency * upsamplingFactor_) * (peakOffset / config_.centerFrequency) - signalStartOffset * upsamplingFactor_;
	signalLength_ = origSignalLength * upsamplingFactor_;
	LOG_DEBUG << "signalOffset_: " << signalOffset_ << " signalLength_: " << signalLength_;

	PrepareDataThreadData<MFloat> prepareDataThreadData;
	if (upsamplingFactor_ > 1) {
		prepareDataThreadData.interpolator.prepare(upsamplingFactor_, UPSAMP_FILTER_HALF_TRANSITION_WIDTH);
	}
	prepareDataThreadData.signal.resize(signalLength_);
	prepareDataTLS_ = std::make_unique<tbb::enumerable_thread_specific<PrepareDataThreadData<MFloat>>>(prepareDataThreadData);

	data_ = std::make_unique<VectorialCombinedTwoMediumImagingCUDAProcessor2Data>();

	if (Log::isDebugEnabled()) {
		int device;
		exec(hipGetDevice(&device));

		hipDeviceProp_t prop;
		exec(hipGetDeviceProperties(&prop, device));
		LOG_DEBUG << "CUDA device: " << prop.name;
	}
}

VectorialCombinedTwoMediumImagingCUDAProcessor2::~VectorialCombinedTwoMediumImagingCUDAProcessor2()
{
	if (data_) {
		LOG_DEBUG << "~VectorialCombinedTwoMediumImagingCUDAProcessor2";
		try {
			exec(hipFree(data_->gridValueImDev));
			exec(hipHostFree(data_->gridValueIm));
			exec(hipFree(data_->gridValueReDev));
			exec(hipHostFree(data_->gridValueRe));
#ifdef USE_TRANSPOSE
			exec(hipFree(data_->rawDataTDev));
#endif
			for (auto p : data_->rawDataDevList) {
				exec(hipFree(p));
			}
			for (auto p : data_->rawDataList) {
				exec(hipHostFree(p));
			}
			exec(hipFree(data_->rxApodDev));
		} catch (std::exception& e) {
			LOG_ERROR << "[~VectorialCombinedTwoMediumImagingCUDAProcessor2] Error: " << e.what();
		} catch (...) {
			LOG_ERROR << "[~VectorialCombinedTwoMediumImagingCUDAProcessor2] Caught an unknown exception.";
		}
	}
}

void
VectorialCombinedTwoMediumImagingCUDAProcessor2::process(
						const std::vector<StepConfiguration>& stepConfigList,
						const std::vector<XZ<MFloat>>& interfacePointList,
						const std::vector<MFloat>& rxApod,
						const Matrix<XZ<MFloat>>& gridXZ,
						Matrix<std::complex<MFloat>>& gridValue)
{
	//LOG_DEBUG << "BEGIN ========== VectorialCombinedTwoMediumImagingCUDAProcessor2::process ==========";

	if (stepConfigList.empty()) {
		THROW_EXCEPTION(InvalidParameterException, "The list of step configurations is empty.");
	}
	if (gridXZ.n1() != gridValue.n1() || gridXZ.n2() != gridValue.n2()) {
		THROW_EXCEPTION(InvalidParameterException, "gridXZ and gridValue have different sizes.");
	}

	const std::size_t samplesPerChannelLow = acqDataList_[0].n2();

	minRowIdx_.resize(gridXZ.n1() /* number of columns */);
	firstGridPointIdx_.resize(gridXZ.n1() /* number of columns */);
	delayTensor_.resize(gridXZ.n1() /* number of columns */, gridXZ.n2() /* number of rows */, config_.numElementsMux);
	signalTensor_.resize(stepConfigList.size(), config_.numElements, signalLength_);
	medium1DelayMatrix_.resize(config_.numElementsMux, interfacePointList.size());

	XZ<MFloat> p1 = interfacePointList[0];
	XZ<MFloat> p2 = interfacePointList[1];
	const MFloat dx = p2.x - p1.x;
	const MFloat dz = p2.z - p1.z;
	const MFloat r = std::sqrt(dx * dx + dz * dz);
	const unsigned int fermatBlockSize = FermatPrinciple::calcBlockSizeForTwoStepSearch(interfacePointList.size(), r, lambda2_, maxFermatBlockSize_);
	LOG_DEBUG << "fermatBlockSize: " << fermatBlockSize;

#ifdef USE_EXECUTION_TIME_MEASUREMENT
	Timer minRowIdxTimer;
#endif
	//==================================================
	// Find the z-coordinate of the interface in
	// each column (minimum row).
	//==================================================
	const MFloat zStepGrid = gridXZ(0, 1).z - gridXZ(0, 0).z;
	unsigned int gridPointIdx = 0;
	for (unsigned int col = 0; col < gridXZ.n1(); ++col) {
		auto& point = gridXZ(col, 0);
		firstGridPointIdx_[col] = gridPointIdx; // the points below the interface are not considered

		// Find the z coordinate of the interface.
		MFloat zIdxMin;
		unsigned int idxMin;
		FermatPrinciple::findNearestPointInXInTwoSteps(
				fermatBlockSize,
				interfacePointList,
				point.x,
				zIdxMin, idxMin);

		if (zIdxMin <= gridXZ(col, 0).z) {
			minRowIdx_[col] = 1;
		} else if (zIdxMin >= gridXZ(col, gridXZ.n2() - 1).z) {
			minRowIdx_[col] = gridXZ.n2(); // after the last index
		} else {
			minRowIdx_[col] = static_cast<unsigned int>(std::ceil((zIdxMin - gridXZ(col, 0).z) / zStepGrid)) + 1;
		}
		if (minRowIdx_[col] >= gridXZ.n2()) {
			THROW_EXCEPTION(InvalidValueException, "No valid rows in column " << col << '.');
		}

		gridPointIdx += gridXZ.n2() - minRowIdx_[col];
	}
	LOG_DEBUG << "number of valid grid points: " << gridPointIdx;
#ifdef USE_EXECUTION_TIME_MEASUREMENT
	tMinRowIdx.put(minRowIdxTimer.getTime());
#endif
	const unsigned int cols = gridXZ.n1();
	if (cols == 0) {
		THROW_EXCEPTION(InvalidValueException, "Zero columns in the grid.");
	}
	std::size_t pointSum = 0;
	for (unsigned int col = 0; col < cols; ++col) {
		pointSum += gridXZ.n2() - minRowIdx_[col];
	}
	const std::size_t numGridPoints = pointSum;
	LOG_DEBUG << "cols: " << cols << " numGridPoints: " << numGridPoints;

#ifdef USE_TRANSPOSE
	const std::size_t transpNumGridPoints = roundUpToMultipleOfGroupSize(numGridPoints, TRANSP_BLOCK_SIZE);
	LOG_DEBUG << "numGridPoints: " << numGridPoints << " transpNumGridPoints: " << transpNumGridPoints;
	rawDataN1_ = transpNumGridPoints;
	rawDataN2_ = 2 * config_.numElements /* real, imag */;
#else
	rawDataN1_ = 2 * config_.numElements /* real, imag */;
	rawDataN2_ = numGridPoints;
#endif
	rawDataSizeInBytes_ = rawDataN1_ * rawDataN2_ * sizeof(MFloat);

	if (!data_->cudaDataInitialized) {
		exec(hipMalloc(&data_->rxApodDev, Util::sizeInBytes(rxApod)));
		for (auto& p : data_->rawDataList) {
			exec(hipHostMalloc(&p, rawDataSizeInBytes_));
		}
		for (auto& p : data_->rawDataDevList) {
			exec(hipMalloc(&p, rawDataSizeInBytes_));
		}
#ifdef USE_TRANSPOSE
		exec(hipMalloc(&data_->rawDataTDev, rawDataSizeInBytes_));
#endif
		exec(hipHostMalloc(&data_->gridValueRe, numGridPoints * sizeof(MFloat)));
		exec(hipMalloc(&data_->gridValueReDev, numGridPoints * sizeof(MFloat)));
		exec(hipHostMalloc(&data_->gridValueIm, numGridPoints * sizeof(MFloat)));
		exec(hipMalloc(&data_->gridValueImDev, numGridPoints * sizeof(MFloat)));
		data_->cudaDataInitialized = true;
	}

	// Prepare buffers.
	exec(hipMemset(data_->gridValueReDev, 0, numGridPoints * sizeof(MFloat)));
	exec(hipMemset(data_->gridValueImDev, 0, numGridPoints * sizeof(MFloat)));
	exec(hipMemcpy(data_->rxApodDev, rxApod.data(), Util::sizeInBytes(rxApod), hipMemcpyHostToDevice));

	const MFloat c2ByC1 = config_.propagationSpeed2 / config_.propagationSpeed1;

	ArrayGeometry::getElementX2D(config_.numElementsMux, config_.pitch, MFloat(0) /* offset */, xArray_);

#ifdef USE_EXECUTION_TIME_MEASUREMENT
	Timer medium1DelayMatrixTimer;
#endif
	for (unsigned int elem = 0; elem < config_.numElementsMux; ++elem) {
		MFloat* delays = &medium1DelayMatrix_(elem, 0);
		for (unsigned int i = 0; i < interfacePointList.size(); ++i) {
			const XZ<MFloat>& ifPoint = interfacePointList[i];
			delays[i] = Geometry::distance2DY0(xArray_[elem], ifPoint.x, ifPoint.z) * c2ByC1;
		}
	}
#ifdef USE_EXECUTION_TIME_MEASUREMENT
	tMedium1DelayMatrix.put(medium1DelayMatrixTimer.getTime());

	Timer calculateDelaysTimer;
#endif
	CalculateDelays2<MFloat> calculateDelaysOp = {
		gridXZ.n2(),
		config_,
		config_.samplingFrequency * upsamplingFactor_,
		config_.samplingFrequency * upsamplingFactor_ / config_.propagationSpeed2,
		1 / config_.propagationSpeed1,
		1 / config_.propagationSpeed2,
		fermatBlockSize,
		interfacePointList,
		xArray_,
		minRowIdx_,
		medium1DelayMatrix_,
		gridXZ,
		delayTensor_
	};
	//tbb::parallel_for(tbb::blocked_range<unsigned int>(0, gridXZ.n1()), calculateDelaysOp);
	tbb::parallel_for(tbb::blocked_range<unsigned int>(0, gridXZ.n1(), 1 /* grain size */), calculateDelaysOp, tbb::simple_partitioner());
	//calculateDelaysOp(tbb::blocked_range<unsigned int>(0, gridXZ.n1())); // single-thread
#ifdef USE_EXECUTION_TIME_MEASUREMENT
	tCalculateDelays.put(calculateDelaysTimer.getTime());
#endif
	// Only one transmit element.
#ifdef USE_EXECUTION_TIME_MEASUREMENT
	Timer prepareDataTimer;
#endif
	unsigned int stepIdx = 0;
	for (const auto& stepConfig : stepConfigList) {
		PrepareDataWithOneTxElem2<MFloat> prepareDataOp = {
			samplesPerChannelLow,
			acqDataList_,
			upsamplingFactor_,
			stepIdx,
			stepConfig.baseElemIdx,
			*prepareDataTLS_,
			signalTensor_
		};
		//tbb::parallel_for(tbb::blocked_range<unsigned int>(0, config_.numElements), prepareDataOp);
		tbb::parallel_for(tbb::blocked_range<unsigned int>(0, config_.numElements, 1 /* grain size */), prepareDataOp, tbb::simple_partitioner());
		//prepareDataOp(tbb::blocked_range<unsigned int>(0, config_.numElements)); // single-thread

		++stepIdx;
	}
#ifdef USE_EXECUTION_TIME_MEASUREMENT
	tPrepareData.put(prepareDataTimer.getTime());

	Timer processColumnTimer;
#endif
	std::size_t procImageKernelGlobalSize = roundUpToMultipleOfGroupSize(numGridPoints, BLOCK_SIZE);
	LOG_DEBUG << numGridPoints << ':' << procImageKernelGlobalSize << ':' << BLOCK_SIZE;

	//==================================================
	// Step configuration loop.
	//==================================================
	for (unsigned int i = 0; i < stepConfigList.size(); ++i) {
		const unsigned int rawBufferIdx = i % data_->rawDataList.size();
		const auto& stepConfig = stepConfigList[i];
		LOG_DEBUG << "stepConfig.baseElemIdx: " << stepConfig.baseElemIdx << " rawBufferIdx: " << rawBufferIdx;

		if (data_->rawDataList.size() == 1) {
			exec(hipDeviceSynchronize());
		}

		Timer delayStoreTimer;

		//==================================================
		// Delay and store.
		//==================================================
		ProcessColumnWithOneTxElem2<MFloat> processColumnOp = {
			static_cast<unsigned int>(gridXZ.n2()),
			0,
			config_,
			signalOffset_,
			signalTensor_,
			stepConfig,
			minRowIdx_,
			firstGridPointIdx_,
			delayTensor_,
			data_->rawDataList[rawBufferIdx],
			rawDataN2_,
		};

		//tbb::parallel_for(tbb::blocked_range<unsigned int>(0, cols), processColumnOp);
		tbb::parallel_for(tbb::blocked_range<unsigned int>(0, cols, 1 /* grain size */), processColumnOp, tbb::simple_partitioner());
		//processColumnOp(tbb::blocked_range<unsigned int>(0, cols)); // single-thread
		LOG_DEBUG << "DELAY-STORE " << delayStoreTimer.getTime();

		if (data_->rawDataList.size() > 1) {
			exec(hipDeviceSynchronize());
		}

		exec(hipMemcpyAsync(data_->rawDataDevList[rawBufferIdx], data_->rawDataList[rawBufferIdx], rawDataSizeInBytes_, hipMemcpyHostToDevice));

#ifdef USE_TRANSPOSE
		{
			dim3 gridDim(rawDataN2_ / TRANSP_BLOCK_SIZE, rawDataN1_ / TRANSP_BLOCK_SIZE);
			dim3 blockDim(TRANSP_BLOCK_SIZE, TRANSP_BLOCK_SIZE);
			const unsigned int sharedMemSize = TRANSP_BLOCK_SIZE * TRANSP_BLOCK_SIZE * sizeof(MFloat);

			transposeKernel<<<gridDim, blockDim, sharedMemSize>>>(
							data_->rawDataDevList[rawBufferIdx],
							data_->rawDataTDev,
							rawDataN2_,
							rawDataN1_);
			checkKernelLaunchError();
		}
#endif
		if (coherenceFactor_.enabled()) {
			std::vector<MFloat> cfConstants;
			coherenceFactor_.implementation().getConstants(cfConstants);

			processImagePCFKernel<<<procImageKernelGlobalSize / BLOCK_SIZE, BLOCK_SIZE>>>(
#ifdef USE_TRANSPOSE
							data_->rawDataTDev,
							rawDataN1_,
#else
							data_->rawDataDevList[rawBufferIdx],
							rawDataN2_,
#endif
							data_->gridValueReDev,
							data_->gridValueImDev,
							data_->rxApodDev,
							cfConstants[2] /* factor */);
			checkKernelLaunchError();
		} else {
			processImageKernel<<<procImageKernelGlobalSize / BLOCK_SIZE, BLOCK_SIZE>>>(
#ifdef USE_TRANSPOSE
							data_->rawDataTDev,
							rawDataN1_,
#else
							data_->rawDataDevList[rawBufferIdx],
							rawDataN2_,
#endif
							data_->gridValueReDev,
							data_->gridValueImDev,
							data_->rxApodDev);
			checkKernelLaunchError();
		}
	}

	exec(hipMemcpy(data_->gridValueRe, data_->gridValueReDev, numGridPoints * sizeof(MFloat), hipMemcpyDeviceToHost));
	exec(hipMemcpy(data_->gridValueIm, data_->gridValueImDev, numGridPoints * sizeof(MFloat), hipMemcpyDeviceToHost));

	//==================================================
	// Read the formed image.
	//==================================================
	for (unsigned int col = 0; col < cols; ++col) {
		for (unsigned int row = 0; row < minRowIdx_[col]; ++row) {
			gridValue(col, row) = 0;
		}
		unsigned int gridPointIdx = firstGridPointIdx_[col];
		for (unsigned int row = minRowIdx_[col]; row < gridXZ.n2(); ++row, ++gridPointIdx) {
			gridValue(col, row) = std::complex<MFloat>(
							data_->gridValueRe[gridPointIdx],
							data_->gridValueIm[gridPointIdx]);
		}
	}

#ifdef USE_EXECUTION_TIME_MEASUREMENT
	tProcessColumn.put(processColumnTimer.getTime());
#endif
	//LOG_DEBUG << "END ========== VectorialCombinedTwoMediumImagingCUDAProcessor2::process ==========";
}

} // namespace Lab
